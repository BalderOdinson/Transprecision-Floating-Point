#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include "tensor_lib.h"
#include "hipblas.h"
#include "hiprand.h"
#include <chrono>
#include "convolution_extensions.h"

#define NEW_LINE "\n"
#define DOUBLE_NEW_LINE "\n\n"

template<typename T>
void diff_checker(transprecision_floating_point::tensor<T> const& first, transprecision_floating_point::tensor<T> const& second)
{
	size_t diff = 0;
	for (size_t i = 0; i < first.shape()[0]; ++i)
	{
		for (size_t j = 0; j < first.shape()[1]; ++j)
		{
			float f = first[transprecision_floating_point::tensor_shape({ i,j })];
			float s = second[transprecision_floating_point::tensor_shape({ i,j })];
			if (fabs(f - s) > 0.00001f)
			{
				std::cout << "Diff at index: (" << std::to_string(i) + "," + std::to_string(j) << ") Values: (" << std::to_string(f) + "," + std::to_string(s) << ")\n";
				++diff;
			}
		}
	}

	std::cout << "Differences found: " << diff << "\n";
}

//int main()
//{
//	// Choose which GPU to run on, change this on a multi-GPU system.
//	auto cudaStatus = hipSetDevice(0);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//		return 1;
//	}
//
//	transprecision_floating_point::tensor_lib_init::init();
//
//	hipblasHandle_t handle = transprecision_floating_point::tensor_lib_init::cublas_handle();
//
//	try
//	{
//		transprecision_floating_point::random_engine::set_seed(100);
//		auto a = transprecision_floating_point::tensor<float>::random(transprecision_floating_point::tensor_shape{ 1000, 1000 }, transprecision_floating_point::normal_distribution());
//		auto b = transprecision_floating_point::tensor<float>::random(transprecision_floating_point::tensor_shape{ 1000, 1000 }, transprecision_floating_point::normal_distribution());
//		//transprecision_floating_point::tensor<float> c(transprecision_floating_point::tensor_shape{ 60000,750 });
//
//		double avg_time_blas = 0;
//		double avg_time = 0;
//		for (auto i = 0; i < 100; ++i)
//		{
//			{
//				auto const start_time = std::chrono::high_resolution_clock::now();
//				{
//					auto result = b.argmax(1);
//				}
//
//				auto const end_time = std::chrono::high_resolution_clock::now();
//				auto time = std::chrono::duration<float, std::chrono::milliseconds::period>(end_time - start_time).count();
//				std::cout << "Time passed: " <<  time << "ms" << std::endl;
//				avg_time_blas += time;
//			}
//
//			{
//				/*auto const start_time = std::chrono::high_resolution_clock::now();
//				{
//					auto result = a.argmax(1);
//				}
//				auto const end_time = std::chrono::high_resolution_clock::now();
//				auto time = std::chrono::duration<float, std::chrono::milliseconds::period>(end_time - start_time).count();
//				std::cout << "Time passed: " << time << "ms" << std::endl;
//				avg_time += time;*/
//			}
//		}
//
//		std::cout << "Avg time 60000: " << avg_time_blas / 100 << NEW_LINE;
//		//std::cout << "Avg time 1000: " << avg_time / 100 << NEW_LINE;
//
//		//std::fstream file("output.txt", std::fstream::out);
//		//std::fstream ex_file("expected_output.txt", std::fstream::out);
//		//auto a = transprecision_floating_point::tensor<float>::random(transprecision_floating_point::tensor_shape{ 4,2 }, transprecision_floating_point::normal_distribution());
//		//auto b = transprecision_floating_point::tensor<float>::random(transprecision_floating_point::tensor_shape{ 2,3 }, transprecision_floating_point::normal_distribution());;
//		//auto y = transprecision_floating_point::tensor<float>::random(transprecision_floating_point::tensor_shape{ 1,4 }, transprecision_floating_point::normal_distribution());;
//		////auto result = a.dot(~a);
//
//		//file << a << DOUBLE_NEW_LINE << a.argmax(1);
//
//		//transprecision_floating_point::tensor_extensions::gemm_ex(1, a, false, b, false, 1,true, c);
//		////transprecision_floating_point::tensor_extensions::gemv(1.f, a, true, b, true, 1.f, true, c);
//		//
//		///*file <<
//		//	~a << DOUBLE_NEW_LINE <<
//		//	~b << DOUBLE_NEW_LINE <<
//		//	~y << DOUBLE_NEW_LINE <<
//		//	~a * ~b << DOUBLE_NEW_LINE <<
//		//	c << DOUBLE_NEW_LINE;*/
//
//		//auto res = (a.dot(b)) + ~y;
//
//		//diff_checker(res, c);
//
//		/*file << c;
//		ex_file << (~a * ~b) + ~y;*/
//
//		/*transprecision_floating_point::tensor_extensions::axpy(-2.f, c, c);
//		file << DOUBLE_NEW_LINE << c;*/
//		/*file << a << DOUBLE_NEW_LINE << a.sum() << DOUBLE_NEW_LINE << a.argmax(0, false) << DOUBLE_NEW_LINE << a.argmin(1, false) << DOUBLE_NEW_LINE << a.argmax(2, false);*/
//
//		/*auto a = transprecision_floating_point::tensor<float>::random(transprecision_floating_point::tensor_shape{ 2,2,2,2 }, transprecision_floating_point::normal_distribution());
//		auto b = transprecision_floating_point::tensor<float>::random(transprecision_floating_point::tensor_shape{ 2,2,1 }, transprecision_floating_point::normal_distribution());
//		auto c = transprecision_floating_point::tensor<float>::random(transprecision_floating_point::tensor_shape{ 1,2,2 }, transprecision_floating_point::normal_distribution());
//		auto d = 5.f;
//
//		std::fstream file("output.txt", std::fstream::out);
//		file << "MATRIX A -----------------------------------------------------" << NEW_LINE << a << NEW_LINE << "--------------------------------------------------------------" << DOUBLE_NEW_LINE
//			<< "MATRIX A+A ---------------------------------------------------" << NEW_LINE << a + a << NEW_LINE << "--------------------------------------------------------------" << DOUBLE_NEW_LINE
//			<< "MATRIX A -----------------------------------------------------" << NEW_LINE << a << NEW_LINE << "--------------------------------------------------------------" << DOUBLE_NEW_LINE
//			<< "MATRIX B -----------------------------------------------------" << NEW_LINE << b << NEW_LINE << "--------------------------------------------------------------" << DOUBLE_NEW_LINE
//			<< "MATRIX A+B ---------------------------------------------------" << NEW_LINE << a + b << NEW_LINE << "--------------------------------------------------------------" << DOUBLE_NEW_LINE
//			<< "MATRIX A -----------------------------------------------------" << NEW_LINE << a << NEW_LINE << "--------------------------------------------------------------" << DOUBLE_NEW_LINE
//			<< "MATRIX C -----------------------------------------------------" << NEW_LINE << c << NEW_LINE << "--------------------------------------------------------------" << DOUBLE_NEW_LINE
//			<< "MATRIX A+C ---------------------------------------------------" << NEW_LINE << a + c << NEW_LINE << "--------------------------------------------------------------" << DOUBLE_NEW_LINE
//			<< "MATRIX A -----------------------------------------------------" << NEW_LINE << a << NEW_LINE << "--------------------------------------------------------------" << DOUBLE_NEW_LINE
//			<< "MATRIX A+D ---------------------------------------------------" << NEW_LINE << a + d << NEW_LINE << "--------------------------------------------------------------";*/
//	}
//	catch (std::exception const& exc)
//	{
//		std::cerr << exc.what() << NEW_LINE;
//	}
//
//	transprecision_floating_point::tensor_lib_init::destroy();
//
//	// hipDeviceReset must be called before exiting in order for profiling and
//	// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	cudaStatus = hipDeviceReset();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceReset failed!");
//		return 1;
//	}
//
//
//
//	return 0;
//}
