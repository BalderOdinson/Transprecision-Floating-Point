#include "tensor_lib_init.h"


namespace transprecision_floating_point
{
	void tensor_lib_init::init(hipblasHandle_t cublas_handle, hipdnnHandle_t cudnn_handle)
	{
		if (cublas_handle)
		{
			cublas_handle_() = cublas_handle;
			return;
		}

		if(cudnn_handle)
		{
			cudnn_handle_() = cudnn_handle;
			return;
		}

		CHECK_CUBLAS_ERROR(hipblasCreate(&cublas_handle_()));
		CHECK_CUDNN_ERROR(hipdnnCreate(&cudnn_handle_()));
	}

	hipblasHandle_t tensor_lib_init::cublas_handle()
	{
		return cublas_handle_();
	}

	hipdnnHandle_t tensor_lib_init::cudnn_handle()
	{
		return cudnn_handle_();
	}

	void tensor_lib_init::destroy()
	{
		hipblasDestroy(cublas_handle_());
		hipdnnDestroy(cudnn_handle_());
	}

	hipblasHandle_t& tensor_lib_init::cublas_handle_()
	{
		static hipblasHandle_t handle = nullptr;
		return handle;
	}

	hipdnnHandle_t& tensor_lib_init::cudnn_handle_()
	{
		static hipdnnHandle_t handle = nullptr;
		return handle;
	}
}
