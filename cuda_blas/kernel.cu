#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "tensor_tests.h"
#include <fstream>
#include <chrono>
#include "distributions.h"
#include "../flexfloat_cuda/flexfloat_cuda.h"
using float32 = transprecision_floating_point::cuda::flexfloat_cuda<8, 23>;
using float16 = transprecision_floating_point::cuda::flexfloat_cuda<5, 10>;
using float16alt = transprecision_floating_point::cuda::flexfloat_cuda<8, 7>;
using float8 = transprecision_floating_point::cuda::flexfloat_cuda<5, 2>;

int main()
{
	auto cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	try
	{
		/*double avg_time = 0;
		auto t = transprecision_floating_point::cuda_blas::tensor<float8>::create_random(500, 100, transprecision_floating_point::cuda_blas::normal_distribution());
		for (auto i = 0; i < 100; ++i)
		{

			auto const start_time = std::chrono::high_resolution_clock::now();
			auto sum = t.max<1>();
			auto const end_time = std::chrono::high_resolution_clock::now();
			auto time = std::chrono::duration<float, std::chrono::milliseconds::period>(end_time - start_time).count();
			std::cout << "Time passed: " << time << "ms" << std::endl;
			avg_time += time;
		}

		std::cout << "Avg time passed: " << avg_time / 100 << "ms" << std::endl;*/

		/*auto t = transprecision_floating_point::cuda_blas::tensor<float8>::create_random(100, 100, transprecision_floating_point::cuda_blas::normal_distribution());
		transprecision_floating_point::cuda_blas::tensor<float8> t_r({ 100,1 }, float8(1.0));
		std::fstream file("output.txt", std::fstream::out);
		file << t << "\n\n";
		file << produce(t, t_r, [] __device__(float8 first, float8 second) { return first + second; });*/
		/*auto t = transprecision_floating_point::cuda_blas::tensor<float>::create_random(10, 5, transprecision_floating_point::cuda_blas::normal_distribution());
		auto t_1 = transprecision_floating_point::cuda_blas::tensor<float>::create_random(10, 5, transprecision_floating_point::cuda_blas::normal_distribution());
		
		std::fstream file("output.txt", std::fstream::out);
		file << t << "\n\n";
		file << transpose(t) << "\n\n";
		file << t_1 << "\n\n";
		file << transpose(t_1);*/
		transprecision_floating_point::cuda_blas::test_tensor();
	}
	catch (std::exception const& exc)
	{
		std::cerr << exc.what() << "\n";
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	//transprecision_floating_point::cuda_blas::test_tensor();
	return 0;
}
